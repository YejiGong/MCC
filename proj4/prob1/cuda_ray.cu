
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA 0
#define OPENMP 1
#define SPHERES 20

#define rnd(x) (x * hiprand_uniform(&local_state))
#define INF 2e10f
#define DIM 2048

struct Sphere {
    float r, b, g;
    float radius;
    float x, y, z;
    __device__ float hit(float ox, float oy, float* n) const {
        float dx = ox - x;
        float dy = oy - y;
        if (dx * dx + dy * dy < radius * radius) {
            float dz = sqrtf(radius * radius - dx * dx - dy * dy);
            *n = dz / sqrtf(radius * radius);
            return dz + z;
        }
        return -INF;
    }
};

__global__ void initializeSpheres(Sphere* s) {
    int i = threadIdx.x;
    if (i < SPHERES) {
        hiprandState local_state;
        hiprand_init(clock64(), i, 0, &local_state);
        s[i].r = rnd(1.0f);
        s[i].g = rnd(1.0f);
        s[i].b = rnd(1.0f);
        s[i].x = rnd(2000.0f) - 1000;
        s[i].y = rnd(2000.0f) - 1000;
        s[i].z = rnd(2000.0f) - 1000;
        s[i].radius = rnd(200.0f) + 40;
    }
}

__global__ void kernel(Sphere* s, unsigned char* ptr) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    int offset = x + y * DIM;

    float ox = (x - DIM / 2);
    float oy = (y - DIM / 2);

    float r = 0, g = 0, b = 0;
    float maxz = -INF;
    for (int i = 0; i < SPHERES; i++) {
        float n;
        float t = s[i].hit(ox, oy, &n);
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[offset * 4 + 0] = (int)(r * 255);
    ptr[offset * 4 + 1] = (int)(g * 255);
    ptr[offset * 4 + 2] = (int)(b * 255);
    ptr[offset * 4 + 3] = 255;
}

void ppm_write(unsigned char* bitmap, int xdim, int ydim, FILE* fp) {
    int i, x, y;
    fprintf(fp, "P3\n");
    fprintf(fp, "%d %d\n", xdim, ydim);
    fprintf(fp, "255\n");
    for (y = 0; y < ydim; y++) {
        for (x = 0; x < xdim; x++) {
            i = x + y * xdim;
            fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
        }
        fprintf(fp, "\n");
    }
}

int main(int argc, char* argv[]) {
    int no_threads;
    int option;
    int x, y;
    unsigned char* bitmap;
    unsigned char* d_bitmap;

    srand(time(NULL));

    if (argc != 3) {
        printf("> a.out [option] [filename.ppm]\n");
        printf("[option] 0: CUDA, 1~16: OpenMP using 1~16 threads\n");
        printf("for example, '> a.out 8 result.ppm' means executing OpenMP with 8 threads\n");
        exit(0);
    }
    FILE* fp = fopen(argv[2], "w");

    if (strcmp(argv[1], "0") == 0)
        option = CUDA;
    else {
        option = OPENMP;
        no_threads = atoi(argv[1]);
    }

    

    Sphere* temp_s = (Sphere*)malloc(sizeof(Sphere) * SPHERES);

    Sphere* d_temp_s;
    hipMalloc((void**)&d_temp_s, sizeof(Sphere) * SPHERES);
    hipMemcpy(d_temp_s, temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice);

    bitmap = (unsigned char*)malloc(sizeof(unsigned char) * DIM * DIM * 4);
    hipMalloc((void**)&d_bitmap, sizeof(unsigned char) * DIM * DIM * 4);

    dim3 blocks(DIM/16, DIM/16);
    dim3 threads(16,16);


    clock_t start = clock();
    initializeSpheres<<<1, SPHERES>>>(d_temp_s);
    hipMemcpy(temp_s, d_temp_s, sizeof(Sphere) * SPHERES, hipMemcpyDeviceToHost);

    kernel<<<blocks, threads>>>(d_temp_s, d_bitmap);
    hipMemcpy(bitmap, d_bitmap, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyDeviceToHost);

    ppm_write(bitmap, DIM, DIM, fp);

    clock_t end = clock();
    double elapsedTime = (double)(end-start)/CLOCKS_PER_SEC;
    printf("CUDA ray tracing: %.10lf sec\n",elapsedTime);
    printf("[%s] was generated.", argv[2]);
    
    fclose(fp);
    free(bitmap);
    free(temp_s);
    hipFree(d_temp_s);
    hipFree(d_bitmap);

    return 0;
}
