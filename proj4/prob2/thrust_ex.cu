#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>

#define BLOCK_SIZE 256

struct calculate_sum : public thrust::unary_function<long, double>
{
    const double step;

    calculate_sum(double _step) : step(_step) {}

    __host__ __device__
    double operator()(const long& i) const
    {
        double x = (i + 0.5) * step;
        return 4.0 / (1.0 + x * x);
    }
};

int main()
{
    long num_steps = 1000000000;
    double step = 1.0 / (double)num_steps;

    thrust::device_vector<long> d_indices(num_steps);
    thrust::sequence(d_indices.begin(), d_indices.end());

    clock_t start = clock();

    double sum = thrust::transform_reduce(d_indices.begin(), d_indices.end(), calculate_sum(step), 0.0, thrust::plus<double>());

    double pi = step * sum;

    clock_t end = clock();
    double elapsedTime = (double)(end-start)/CLOCKS_PER_SEC;

    printf("Execution Time: %.10lf sec\n", elapsedTime);
    printf("pi=%.10lf\n", pi);

    return 0;
}
